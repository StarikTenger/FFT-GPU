#include "hip/hip_runtime.h"
#include "fft_gpu.h"
#include "util.h"
#include "fl.h"
#include "bitrev_reorder.h"

#include <cstdlib>
#include <iostream>
#include <math.h>
#include <numbers>
#include <thrust/complex.h>
#include <chrono>

using namespace std;


/* Questions:
- In which memory are we working?
- Access is uncoalesced, how to fix it?
*/



__global__ void fft_step(const fl *buff_in, fl *buff_out, size_t N, size_t stride) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idx_1 = i;
    size_t idx_2 = (i ^ (stride / 2)) % N;
    
    if ((i % stride) >= stride / 2) {
        // Swap
        size_t temp = idx_1;
        idx_1 = idx_2;
        idx_2 = temp;
    }
    
    thrust::complex<fl> w = thrust::pow(
        M_E,
        thrust::complex<fl>(0, -2. * M_PI * i / stride)
    );

    thrust::complex<fl> res =
        thrust::complex<fl>(buff_in[idx_1], buff_in[idx_1 + N]) +
        thrust::complex<fl>(buff_in[idx_2], buff_in[idx_2 + N]) * w;



    buff_out[i] = res.real();
    buff_out[i + N] = res.imag();
}

__global__ void conjugate(const fl *buff_in, fl *buff_out, size_t N) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    buff_out[i] = buff_in[i];
    buff_out[i + N] = -buff_in[i + N];
}

__global__ void divide_N(const fl *buff_in, fl *buff_out, size_t N) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    buff_out[i] = buff_in[i] / N;
    buff_out[i + N] = buff_in[i + N] / N;
}

void fft_gpu(const fl *buff_in, fl *buff_out, size_t N) {
    fl *buff1 = new fl[N * 2];
    fl *buff_to_delete = buff1;
    memcpy((void*)buff1, (void*)buff_in, (N * 2) * sizeof(fl));
    fl *buff2 = buff_out;
    

    // Reorder buffer
    bitrev_reorder(buff1, buff2, N);
    swap(buff1, buff2);

    // Allocate buffers on GPU
    fl *buff_gpu1;
    fl *buff_gpu2;
    const size_t buff_size = N * 2 * sizeof(fl);

    hipError_t err;

    err = hipMalloc((void**)&buff_gpu1, buff_size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory for buff_gpu1: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&buff_gpu2, buff_size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory for buff_gpu2: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(buff_gpu1, buff1, buff_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying memory to buff_gpu1: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Define workspace topology
    size_t block_size = 32;
	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(N / block_size, 1);

    for (size_t stride = 2; stride <= N; stride <<= 1) {

        fft_step<<<dimGrid, dimBlock>>>(buff_gpu1, buff_gpu2, N, stride);
        hipDeviceSynchronize();

        swap(buff_gpu1, buff_gpu2);
    }

    hipMemcpy(buff_out, buff_gpu1, buff_size, hipMemcpyDeviceToHost);

    delete[] buff_to_delete;
}



void reverse_fft_gpu(const fl *buff_in, fl *buff_out, size_t N) {
    fl *buff1 = new fl[N * 2];
    fl *buff_to_delete = buff1;
    memcpy((void*)buff1, (void*)buff_in, (N * 2) * sizeof(fl));
    fl *buff2 = buff_out;
    

    // Reorder buffer
    bitrev_reorder(buff1, buff2, N);
    swap(buff1, buff2);

    // Allocate buffers on GPU
    fl *buff_gpu1;
    fl *buff_gpu2;
    const size_t buff_size = N * 2 * sizeof(fl);

    hipError_t err;

    err = hipMalloc((void**)&buff_gpu1, buff_size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory for buff_gpu1: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&buff_gpu2, buff_size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory for buff_gpu2: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(buff_gpu1, buff1, buff_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying memory to buff_gpu1: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Define workspace topology
    size_t block_size = BLOCK_SIZE;
	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(N / block_size, 1);

    conjugate<<<dimGrid, dimBlock>>>(buff_gpu1, buff_gpu2, N);
    hipDeviceSynchronize();
    swap(buff_gpu1, buff_gpu2);

    for (size_t stride = 2; stride <= N; stride <<= 1) {

        fft_step<<<dimGrid, dimBlock>>>(buff_gpu1, buff_gpu2, N, stride);
        hipDeviceSynchronize();

        swap(buff_gpu1, buff_gpu2);
    }

    conjugate<<<dimGrid, dimBlock>>>(buff_gpu1, buff_gpu2, N);
    hipDeviceSynchronize();
    swap(buff_gpu1, buff_gpu2);

    divide_N<<<dimGrid, dimBlock>>>(buff_gpu1, buff_gpu2, N);
    hipDeviceSynchronize();
    swap(buff_gpu1, buff_gpu2);

    hipMemcpy(buff_out, buff_gpu1, buff_size, hipMemcpyDeviceToHost);

    delete[] buff_to_delete;
}
#include "fl.h"
#include "genbmp.h"
#include "util.h"
#include "fft_cpu.h"
#include "fft_gpu.h"
#include "fft_gpu_shared.h"

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <set>
#include <vector>

using namespace std;


int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <N>" << std::endl;
        return 1;
    }

    size_t N = std::atoi(argv[1]);
    if (N & (N - 1)) {
        std::cerr << "N must be a power of 2" << std::endl;
        return 1;
    }

    fl *buff_in = new fl[N * 2];
    fl *buff_out = new fl[N * 2];

    for (int i = 0; i < N; i++) {
        buff_in[i] = i < N / 2 ? 0 : 1;
        buff_in[i + N] = 0;
    }

    size_t steps = 0;
    for (size_t stride = 2; stride <= N; stride <<= 1) {
        steps++;
    }

    // Initialize graph

    stringstream graph_stream;
    graph_stream << "digraph G {\n";

    for (size_t step = 0; step <= steps; step++) {
        for (size_t i = 0; i < N; i++) {
            graph_stream << step << "." << i << " [pos=\"" << step * 3 << "," << (N - i) << "!\"];\n";
        }
    }

    // ================================== FFT ==================================

    // GPU shared
    {
        cout << "\nRunning fft gpu shared" << endl;


        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        auto chrono_start = chrono::high_resolution_clock::now();


        hipEventRecord(start);

        fft_gpu_shared(buff_in, buff_out, N);

        auto chrono_end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = chrono_end - chrono_start;
        cout << "FFT execution time (chrono): " << elapsed.count() * 1000 << " ms" << endl;

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        std::cout << "Time taken for gpu (shared mem): " << milliseconds << " ms" << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Serialize output
        serialize_output(buff_out, N, "output_cpp_gpu_shared.txt", 6);
    }

    // =========================================================================
}
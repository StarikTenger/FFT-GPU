#include "fl.h"
#include "genbmp.h"
#include "util.h"
#include "fft_cpu.h"
#include "fft_gpu.h"
#include "fft_gpu_shared.h"

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <set>
#include <vector>

using namespace std;


int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <N>" << std::endl;
        return 1;
    }

    size_t N = std::atoi(argv[1]);
    if (N & (N - 1)) {
        std::cerr << "N must be a power of 2" << std::endl;
        return 1;
    }

    fl *buff_in = new fl[N * 2];
    fl *buff_out = new fl[N * 2];

    for (int i = 0; i < N; i++) {
        buff_in[i] = i < N / 2 ? 0 : 1;
        buff_in[i + N] = 0;
    }

    size_t steps = 0;
    for (size_t stride = 2; stride <= N; stride <<= 1) {
        steps++;
    }

    // Initialize graph

    stringstream graph_stream;
    graph_stream << "digraph G {\n";

    for (size_t step = 0; step <= steps; step++) {
        for (size_t i = 0; i < N; i++) {
            graph_stream << step << "." << i << " [pos=\"" << step * 3 << "," << (N - i) << "!\"];\n";
        }
    }

    // ================================== FFT ==================================

    // CPU
    {
        auto start = chrono::high_resolution_clock::now();

        fft_cpu(buff_in, buff_out, N, graph_stream);

        auto end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = end - start;
        cout << "Time taken for cpu: " << elapsed.count() * 1000 << " ms" << endl;

        // Serialize output
        serialize_output(buff_out, N, "output_cpp_seq.txt", 6);
    }

    // GPU
    {
        cout << " --- " << endl;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        auto chrono_start = chrono::high_resolution_clock::now();


        hipEventRecord(start);

        fft_gpu(buff_in, buff_out, N);

        auto chrono_end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = chrono_end - chrono_start;
        cout << "FFT execution time (chrono): " << elapsed.count() * 1000 << " ms" << endl;

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        std::cout << "Time taken for gpu: " << milliseconds << " ms" << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Serialize output
        serialize_output(buff_out, N, "output_cpp_gpu.txt", 6);

    }

    // GPU shared
    {
        cout << " --- " << endl;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        auto chrono_start = chrono::high_resolution_clock::now();


        hipEventRecord(start);

        fft_gpu_shared(buff_in, buff_out, N);

        auto chrono_end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = chrono_end - chrono_start;
        cout << "FFT execution time (chrono): " << elapsed.count() * 1000 << " ms" << endl;

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        std::cout << "Time taken for gpu (shared mem): " << milliseconds << " ms" << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Serialize output
        serialize_output(buff_out, N, "output_cpp_gpu_shared.txt", 6);
    }

    // =========================================================================

    graph_stream << "}";

    // Dump graph to file
    ofstream graph_file("graph.dot");
    graph_file << graph_stream.str();
    graph_file.close();

    // {
    //     // Shared topology test
    //     int log_bsize = 1;
    //     int bsize = 1<<log_bsize;
    //     int N = 16;

    //     for (int epoch = 0; epoch < 4; epoch++) {
    //         cout << "Epoch " << epoch << "\n";

    //         vector<set<pair<int, int>>> mapping(N);

    //         for (int b = 0; b < N / bsize; b++) {
    //             int pos = (b>>(epoch * log_bsize)) * 1<<(log_bsize * (epoch + 1));
    //             int offset = b % (1<<(log_bsize * epoch));
    //             cout << "b = " << b << "; pos = " << pos << "; offset = " << offset << endl;
    //             for (int i = 0; i < bsize; i++) {
    //                 mapping[pos + offset + i * (1<<(log_bsize * epoch))].insert({b, i});
    //             }
    //         }

    //         for (int i = 0; i < N; i++) {
    //             cout << i << "\t: ";
    //             for (auto it = mapping[i].begin(); it != mapping[i].end(); ++it) {
    //                 if (it != mapping[i].begin()) {
    //                     cout << ", ";
    //                 }
    //                 cout << it->first << " : " << it->second;
    //             }
    //             cout << endl;
    //         }
    //         cout << endl;
    //     }      

    // }
}
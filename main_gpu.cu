#include "fl.h"
#include "genbmp.h"
#include "util.h"
#include "fft_gpu.h"

#include <cstdlib>
#include <iostream>
#include <chrono>
#include <set>
#include <vector>

using namespace std;


int main(int argc, char *argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <N>" << std::endl;
        return 1;
    }

    size_t N = std::atoi(argv[1]);
    if (N & (N - 1)) {
        std::cerr << "N must be a power of 2" << std::endl;
        return 1;
    }

    fl *buff_in = new fl[N * 2];
    fl *buff_out = new fl[N * 2];

    for (int i = 0; i < N; i++) {
        buff_in[i] = i < N / 2 ? 0 : 1;
        buff_in[i + N] = 0;
    }

    size_t steps = 0;
    for (size_t stride = 2; stride <= N; stride <<= 1) {
        steps++;
    }

    // Initialize graph

    stringstream graph_stream;
    graph_stream << "digraph G {\n";

    for (size_t step = 0; step <= steps; step++) {
        for (size_t i = 0; i < N; i++) {
            graph_stream << step << "." << i << " [pos=\"" << step * 3 << "," << (N - i) << "!\"];\n";
        }
    }

    // ================================== FFT ==================================


    // GPU
    {
        cerr << "\nRunning fft gpu" << endl;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        auto chrono_start = chrono::high_resolution_clock::now();


        hipEventRecord(start);

        fft_gpu(buff_in, buff_out, N);

        auto chrono_end = chrono::high_resolution_clock::now();
        chrono::duration<double> elapsed = chrono_end - chrono_start;
        cerr << "FFT execution time (chrono): " << elapsed.count() * 1000 << " ms" << endl;

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        std::cerr << "Time taken for gpu: " << milliseconds << " ms" << std::endl;
        //std::cout << "GPU,\t\t" << N << ",\t" << milliseconds << std::endl;
        std::cout << BLOCK_SIZE << ",\t\t" << N << ",\t" << milliseconds << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Serialize output
        serialize_output(buff_out, N, "output_cpp_gpu.txt", 6);

        swap(buff_in, buff_out);
        reverse_fft_gpu(buff_in, buff_out, N);
        serialize_output(buff_out, N, "output_cpp_gpu_reversed.txt", 6);

    }

   
    // =========================================================================
}